/*
 *  Copyright (c) 2022 Bertrand LE GAL
 *
 *  This software is provided 'as-is', without any express or
 *  implied warranty. In no event will the authors be held
 *  liable for any damages arising from the use of this software.
 *
 *  Permission is granted to anyone to use this software for any purpose,
 *  including commercial applications, and to alter it and redistribute
 *  it freely, subject to the following restrictions:
 *
 *  1. The origin of this software must not be misrepresented;
 *  you must not claim that you wrote the original software.
 *  If you use this software in a product, an acknowledgment
 *  in the product documentation would be appreciated but
 *  is not required.
 *
 *  2. Altered source versions must be plainly marked as such,
 *  and must not be misrepresented as being the original software.
 *
 *  3. This notice may not be removed or altered from any
 *  source distribution.
 *
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
/*
 *
 *
 *##############################################################################
 *
 *
 */
#if defined(__NVCC__)
#define THREADGROUP_SIZE 256
#define BSIZE 256 			   // Block size in # of threads
#define softeningSquared 0.01f // original plumer softener is 0.025. here the value is square of it.
#define damping          1.00f // 0.999f
#define ep               0.67f // 0.5f
/*
 *
 *
 *##############################################################################
 *
 *
 */
 __global__ void body2Body_copy(
              float4* c_pos,
        const float4* n_pos,
        const float4* v_dat,
        const int   nElements)
{
    int particule_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    if( particule_id < nElements )
    {
        const float4 newPos   = n_pos[ particule_id ];
        c_pos[ particule_id ] = newPos;
    }
}
/*
 *
 *
 *##############################################################################
 *
 *
 */
__global__ void body2Body_exec(
        const float4* c_pos,
              float4* n_pos,
              float4* v_dat,
        const int     nElements)
{

    extern __shared__ float4 shPosition[THREADGROUP_SIZE];

    const int tid          = threadIdx.x;
    const int particule_id = blockIdx.x * blockDim.x + threadIdx.x;

    //
    // Si jamais a cause des threads/grilles ont deborde...
    //

    if (particule_id < nElements)
    {
        //
        // Dans le cas contraire on commence notre calcul !
        //

        float4 myPosition = c_pos[ particule_id ];

        //
        //
        //

//        float4 myVelocity = v_dat[ particule_id ];
        float3 myAccel = {0.f, 0.f, 0.f};

        for (int idx = 0; idx < nElements; idx += blockDim.x)
        {

            shPosition[tid] = c_pos[idx + tid];

            __syncthreads();

            #pragma unroll 4
            for (uint p = 0; p < blockDim.x; p += 1)
            {

                const float4 other = shPosition[p];
                
                float rx = other.x - myPosition.x;
                float ry = other.y - myPosition.y;
                float rz = other.z - myPosition.z;

                float d = (rx * rx) + (ry * ry) + (rz * rz);
                
                float d3  = 10.0f * other.w;
                float div = (sqrt(d) * d);
                if ( d > 1.0f )
                {
                    d3 /= div; // Multi modifiable
                }

                myAccel.x += rx * d3;
                myAccel.y += ry * d3;
                myAccel.z += rz * d3;
            }

            __syncthreads();

        }

        //
        // update position
        //

        __syncthreads();

        float4 myVelocity = v_dat[ particule_id ];
        myVelocity.x += myAccel.x * 2.0f;
        myVelocity.y += myAccel.y * 2.0f;
        myVelocity.z += myAccel.z * 2.0f;

        myPosition.x = myPosition.x + (myVelocity.x * 0.01f);
        myPosition.y = myPosition.y + (myVelocity.y * 0.01f);
        myPosition.z = myPosition.z + (myVelocity.z * 0.01f);

        __syncthreads();

        //
        // update device memory
        //
        
        n_pos[ particule_id ] = myPosition;     // NEXT POSITION !
        v_dat[ particule_id ] = myVelocity;     // NEW  VELOCITY !
    }

        __syncthreads();
}

#endif